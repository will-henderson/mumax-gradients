#include "hip/hip_runtime.h"
#include "amul.h"
#include "float3.h"
#include <stdint.h>

extern "C" __global__ void
mProp(float* __restrict__ new_lmx, float* __restrict__ new_lmy, float* __restrict__ new_lmz, 
    float* __restrict__ new_lbx, float* __restrict__ new_lby, float* __restrict__ new_lbz,
    float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
    float* __restrict__ bx, float* __restrict__ by, float* __restrict__ bz,
    float* __restrict__ old_lmx, float* __restrict__ old_lmy, float* __restrict__ old_lmz,
    float* __restrict__ alpha_, float alpha_mul, int N) {

        int i = ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
        if (i < N) {
            float3 m = {mx[i], my[i], mz[i]};
            float3 B = {bx[i], by[i], bz[i]};
            float3 l = {old_lmx[i], old_lmy[i], old_lmz[i]};

            float3 lxB = cross(l, B);
            float3 mxB = cross(m, B);
            float3 lxm = cross(l, m);

            float alpha = amul(alpha_, alpha_mul, i);
            float gilb = 1.0f / (1.0f + alpha * alpha);
            float3 m_torque = gilb * (lxB + alpha * (cross(l, mxB) + cross(lxm, B)));
            float3 new_lb = - gilb * (lxm + alpha * cross(lxm, m));

            // we also add the identity because this is from time integration.
            float3 new_lm = m_torque + l;
            new_lmx[i] = new_lm.x;
            new_lmy[i] = new_lm.y;
            new_lmz[i] = new_lm.z;

            new_lbx[i] = new_lb.x;
            new_lby[i] = new_lb.y;
            new_lbz[i] = new_lb.z;

        }
    }
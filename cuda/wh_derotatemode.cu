
#include <hip/hip_runtime.h>
extern "C" __global__ void
derotateMode(float* __restrict__ dstx, float* __restrict__ dsty, float* __restrict__ dstz,
               float* __restrict__  mx,  float* __restrict__  my,
               float* __restrict__  Rxx, float* __restrict__  Rxy, float* __restrict__  Rxz,
               float* __restrict__  Ryx, float* __restrict__  Ryy, float* __restrict__  Ryz,
               int N){

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;

    if(i < N) {  
        dstx[i] = Rxx[i]*mx[i] + Ryx[i]*my[i];
        dsty[i] = Rxy[i]*mx[i] + Ryy[i]*my[i];
        dstz[i] = Rxz[i]*mx[i] + Ryz[i]*my[i];
    }  
    
}
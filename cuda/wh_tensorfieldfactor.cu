
#include <hip/hip_runtime.h>
extern "C" __global__ void
tensorFieldFactor(float* __restrict__  dst,
    float* __restrict__ ms, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;

    if(i < N) {
        if (ms[i] == 0.0f) {
            dst[i] = 0.0f;
        } else {
            dst[i] = - 1.0f / ms[i];
        }
    }
}
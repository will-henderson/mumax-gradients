
#include <hip/hip_runtime.h>
extern "C" __global__ void
rotateMode(float* __restrict__ dstx, float* __restrict__ dsty, 
               float* __restrict__  mx,  float* __restrict__  my,  float* __restrict__  mz,
               float* __restrict__  Rxx, float* __restrict__  Rxy, float* __restrict__  Rxz,
               float* __restrict__  Ryx, float* __restrict__  Ryy, float* __restrict__  Ryz,
               int N){

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;

    if(i < N) {  
        dstx[i] = Rxx[i]*mx[i] + Rxy[i]*my[i] + Rxz[i]*mz[i];
        dsty[i] = Ryx[i]*mx[i] + Ryy[i]*my[i] + Ryz[i]*mz[i];
    }  
    
}

#include <hip/hip_runtime.h>
extern "C" __global__ void
kernmulUniform2D(float* __restrict__ Fxx, float* __restrict__ Fyy, float* __restrict__ Fzz,
                 float* __restrict__ Fxy,
                 float* __restrict__  fftKxx, float* __restrict__  fftKyy, float* __restrict__  fftKzz,
                 float* __restrict__  fftKxy,
                 float* __restrict__ ftu, int Nx, int Ny, int Nz){

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if(ix < Nx && iy < Ny) {
        int I = iy*Nx + ix;

        int e = 2 * I;
        float reftu = ftu[e];       float imftu = ftu[e+1];

        // symmetry factor
        float fxy = 1.0f;
        if (iy > Ny/2) {
            iy = Ny-iy;
            fxy = -fxy;
        }
        I = iy*Nx + ix;

        float Kxx = fftKxx[I];
        float Kyy = fftKyy[I];
        float Kzz = fftKzz[I];
        float Kxy = fxy * fftKxy[I];

        Fxx[e] = Kxx * reftu;       Fxx[e+1] = Kxx * imftu;  
        Fxy[e] = Kxy * reftu;       Fxy[e+1] = Kxy * imftu;
        Fyy[e] = Kyy * reftu;       Fyy[e+1] = Kyy * imftu;
        Fzz[e] = Kzz * reftu;       Fzz[e+1] = Kzz * imftu;


    }


}
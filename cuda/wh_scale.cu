
#include <hip/hip_runtime.h>

extern "C" __global__ void
scale(float* __restrict__  dst,
      float* __restrict__  src, float fac, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;

    if(i < N) {
        dst[i] = fac*src[i];
    }
}

#include <hip/hip_runtime.h>
extern "C" __global__ void
kernmulUniform3D(float* __restrict__ Fxx, float* __restrict__ Fyy, float* __restrict__ Fzz,
                 float* __restrict__ Fyz, float* __restrict__ Fxz, float* __restrict__ Fxy,
                 float* __restrict__  fftKxx, float* __restrict__  fftKyy, float* __restrict__  fftKzz,
                 float* __restrict__  fftKyz, float* __restrict__  fftKxz, float* __restrict__  fftKxy,
                 float* __restrict__ ftu, int Nx, int Ny, int Nz){

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if(ix < Nx && iy < Ny && iz < Nz){
        int I = (iz*Ny + iy)*Nx + ix; 
        int e = 2 * I;
        float reftu = ftu[e];       float imftu = ftu[e+1];

         // fetch kernel

        // minus signs are added to some elements if
        // reconstructed from symmetry.
        float signYZ = 1.0f;
        float signXZ = 1.0f;
        float signXY = 1.0f;

        // use symmetry to fetch from redundant parts:
        // mirror index into first quadrant and set signs.
        if (iy > Ny/2) {
            iy = Ny-iy;
            signYZ = -signYZ;
            signXY = -signXY;
        }
        if (iz > Nz/2) {
            iz = Nz-iz;
            signYZ = -signYZ;
            signXZ = -signXZ;
        }

        // fetch kernel element from non-redundant part
        // and apply minus signs for mirrored parts.
        I = (iz*(Ny/2+1) + iy)*Nx + ix; // Ny/2+1: only half is stored
        float Kxx = fftKxx[I];
        float Kyy = fftKyy[I];
        float Kzz = fftKzz[I];
        float Kyz = fftKyz[I] * signYZ;
        float Kxz = fftKxz[I] * signXZ;
        float Kxy = fftKxy[I] * signXY;

        Fxx[e] = Kxx * reftu;        Fxx[e+1] = Kxx * imftu; 
        Fxy[e] = Kxy * reftu;        Fxy[e+1] = Kxy * imftu;
        Fxz[e] = Kxz * reftu;        Fxz[e+1] = Kxz * imftu;
        Fyy[e] = Kyy * reftu;        Fyy[e+1] = Kyy * imftu;
        Fyz[e] = Kyz * reftu;        Fyz[e+1] = Kyz * imftu;
        Fzz[e] = Kzz * reftu;        Fzz[e+1] = Kzz * imftu;
    }
}
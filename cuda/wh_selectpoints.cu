
#include <hip/hip_runtime.h>
#include <stdint.h>

extern "C" __global__ void 
selectPoints(float* __restrict__ dst, int nSamples, 
             float* __restrict__ src, int Nx, int Ny, int Nz, 
             int32_t* __restrict__ sampleX, int32_t* __restrict__ sampleY, int32_t* __restrict__ sampleZ){

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < nSamples) {
        int e = 2 * i;

        int srcPoint = (sampleZ[i] * Ny + sampleY[i]) * Nx + sampleX[i];
        int srcPointe = 2 * srcPoint;

        dst[e] = src[srcPointe];
        dst[e+1] = src[srcPointe + 1];
    }
}
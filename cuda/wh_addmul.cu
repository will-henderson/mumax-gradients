
#include <hip/hip_runtime.h>
extern "C" __global__ void
addMul(float* __restrict__  dst,
      float* __restrict__  src1, float* src2, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;

    if(i < N) {
        dst[i] += src1[i]*src2[i];
    }
}
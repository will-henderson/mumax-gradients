
#include <hip/hip_runtime.h>
extern "C" __global__ void
complexMagnitude(float* __restrict__ dst, float*__restrict__ src, int N){

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {
        int e = 2*i;
        dst[i] = sqrtf(src[e]*src[e] + src[e+1] * src[e+1]);
    }
}

#include <hip/hip_runtime.h>
extern "C" __global__ void
fourierMode(float* __restrict__ dstReal, float* __restrict__ dstImag,
            float fx, float fy, float fz, int Nx, int Ny, int Nz){

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if(ix < Nx && iy < Ny && iz < Nz){
        int I = (iz*Ny + iy)*Nx + ix;

        float expon = fx * ix + fy * iz + fz * iz;
        dstReal[I] = cosf(expon);
        dstImag[I] = sinf(expon);  
    }        

}

#include <hip/hip_runtime.h>

extern "C" __global__ void
initRotation(float* __restrict__  mx,  float* __restrict__  my,  float* __restrict__  mz,
               float* __restrict__  Rxx, float* __restrict__  Rxy, float* __restrict__  Rxz,
               float* __restrict__  Ryx, float* __restrict__  Ryy, float* __restrict__  Ryz,
               float* __restrict__  Rzx, float* __restrict__  Rzy, float* __restrict__  Rzz,
               int N){

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;

    if(i < N) {
        float Cth = mz[i];
        float Sth = sqrtf(1 - mz[i]*mz[i]);
        float Cph = mx[i] / Sth;
        float Sph = my[i] / Sth;

        Rxx[i] = Cth*Cph; Rxy[i] = Cth*Sph; Rxz[i] = -Sth;
        Ryx[i] = -Sph;    Ryy[i] = Cph;     Ryz[i] = 0.0f;
        Rzx[i] = Sth*Cph; Rzy[i] = Sth*Sph; Rzz[i] = Cth;
    }
}
